#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <sys/resource.h>
extern "C" {
  #include "bmp.h"
}

#define THREADS 16

__global__ void convKernel(int Nfil, int Ncol, unsigned char *d_img, unsigned char *d_out, int win_size, unsigned char *d_win) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int ind = i * Ncol + j;

   //d_out[ind] = 100;//win[win_size * win_size / 2];
  unsigned char *win = (unsigned char* )malloc(win_size * win_size * sizeof(unsigned char));
 // printf("asdf");
  if(win == NULL) {
    d_out[ind] = -1;
    return;
  }
 
  //d_out[ind] = d_img[ind];
  //return;

  if (i < Nfil && j < Ncol) {
    for(int win_i = -win_size/2; win_i <= win_size/2; win_i++) {
      for(int win_j = -win_size/2; win_j <= win_size/2; win_j++) {
        // Si esta fuera, extender el ultimo valor
        int img_i, img_j;
        if(i+win_i < 0)
          img_i = 0;
        else if(i+win_i >= Nfil)
           img_i = Nfil -1;
        else
           img_i = i + win_i;  
        
        if(j+win_j < 0)
          img_j = 0;
        else if(j+win_j >= Ncol)
           img_j = Ncol -1;
        else
           img_j = j + win_j;  

        win[(win_i+win_size/2) * win_size + win_j + win_size/2] = d_img[img_i * Ncol + img_j];
      } 
    }
  // Ordenar valores dentro de la ventana
    int swap;
    for (int c = 0 ; c < win_size * win_size - 1; c++) {
      for (int d = 0 ; d < win_size * win_size - c - 1; d++) {
        if (win[d] > win[d+1]) {
          swap   = win[d];
          win[d]   = win[d+1];
          win[d+1] = swap;
        }
      }
    }
   
   // Retornar la mediana
   if(ind < Nfil * Ncol){
//	printf("yay: %d \n", win[win_size*win_size/2]); 
	 d_out[ind] = win[win_size * win_size / 2];
   }
   free(win);
   return;
  }
}


void InitM(int Nfil, int Ncol, int *v);
void CheckCudaError(char sms[], int line);


int main(int argc, char** argv) {
  unsigned int Nfil, Ncol;
  unsigned int numBytes;
  unsigned int nThreads;
  unsigned int winSize = 3;

  unsigned char *h_img1, *h_img2, *h_img3, *h_img4, *h_out1, *h_out2, *h_out3, *h_out4;
  unsigned char *d_img1, *d_img2, *d_img3, *d_img4, *d_out1, *d_out2, *d_out3, *d_out4, *d_win1, *d_win2, *d_win3, *d_win4;

  bmpInfoHeader* h = (bmpInfoHeader*) malloc(sizeof(bmpInfoHeader));
  unsigned char* img = LoadBMP("test.bmp", h);
  Nfil = h->height;
  Ncol = h->width;
  int imgSize = Ncol * Nfil;
  numBytes = imgSize * sizeof(unsigned char) / 4;

  int count;
  hipGetDeviceCount(&count);
  if(count <  4) { printf("ERROR: no hay suficientes GPUs\n"); exit(0);}  

  hipHostMalloc((unsigned char **)&h_img1, numBytes);
  hipHostMalloc((unsigned char **)&h_img2, numBytes);
  hipHostMalloc((unsigned char **)&h_img3, numBytes);
  hipHostMalloc((unsigned char **)&h_img4, numBytes);
  hipHostMalloc((unsigned char **)&h_out1, numBytes);
  hipHostMalloc((unsigned char **)&h_out2, numBytes);
  hipHostMalloc((unsigned char **)&h_out3, numBytes);
  hipHostMalloc((unsigned char **)&h_out4, numBytes);
  
  if(h_img1 == NULL || h_img2 == NULL || h_img3 == NULL || h_img4 == NULL || h_out1 == NULL || h_out2 == NULL || h_out3 == NULL || h_out4 == NULL) {
    printf("Not enought memory to allocate image");
    exit(0);
  } 
 
  // Inicializar imagen dividida en 4
  int indCol = 0;
  int indFil = 0;
  int baseIndex;
  for(int i = 0; i < imgSize; ++i){
    indCol = i%Ncol;
    indFil = i/Ncol;
    baseIndex = indCol + indFil*Ncol/2;
    if(indCol < Ncol/2 && indFil < Nfil/2){
      h_img1[baseIndex] = img[i*3];
    } else if(indCol >= Ncol/2 && indFil < Nfil/2){
      h_img2[baseIndex-Ncol/2] = img[i*3];
    } else if(indCol < Ncol/2 && indFil >= Nfil/2){
      h_img3[baseIndex-Nfil*Ncol/4] = img[i*3];
    } else if(indCol >= Ncol/2 && indFil >= Nfil/2){
      h_img4[baseIndex-Nfil*Ncol/4-Ncol/2] = img[i*3];
    }
  }

  for(int i = 0; i < 50; ++i){
    printf("%i ",h_img1[i]);
  }
  printf("\n");

  for(int i = 0; i < 50; ++i){
    printf("%i ",h_img2[i]);
  }
  printf("\n");
  for(int i = 0; i < 50; ++i){
    printf("%i ",h_img3[i]);
  }
  printf("\n");
  for(int i = 0; i < 50; ++i){
    printf("%i ",h_img4[i]);
  }
  printf("\n");
  // Obtener Memoria en el device
  hipSetDevice(0);
  hipMalloc((unsigned char**)&d_img1, numBytes); 
  hipMalloc((unsigned char**)&d_out1, numBytes);
  hipMalloc((unsigned char**)&d_win1, winSize*winSize*sizeof(unsigned char));
  
  hipSetDevice(1);
  hipMalloc((unsigned char**)&d_img2, numBytes); 
  hipMalloc((unsigned char**)&d_out2, numBytes); 
  hipMalloc((unsigned char**)&d_win2, winSize*winSize*sizeof(unsigned char));
  
  hipSetDevice(2);
  hipMalloc((unsigned char**)&d_img3, numBytes); 
  hipMalloc((unsigned char**)&d_out3, numBytes); 
  hipMalloc((unsigned char**)&d_win3, winSize*winSize*sizeof(unsigned char));
  
  hipSetDevice(3);
  hipMalloc((unsigned char**)&d_img4, numBytes); 
  hipMalloc((unsigned char**)&d_out4, numBytes); 
  hipMalloc((unsigned char**)&d_win4, winSize*winSize*sizeof(unsigned char));
  CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 

  //
  // KERNEL ELEMENTO a ELEMENTO
  //
  nThreads = THREADS;
  int nBlocksFil = (Nfil/2+nThreads-1)/nThreads;
  int nBlocksCol = (Ncol/2+nThreads-1)/nThreads;

  dim3 dimGridE(nBlocksCol, nBlocksFil, 1);
  dim3 dimBlockE(nThreads, nThreads, 1);


  // Copiar imagen desde el host en el device 
  hipSetDevice(0);
  hipMemcpyAsync(d_img1, h_img1, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
 // hipDeviceSetLimit(hipLimitMallocHeapSize, winSize * winSize * sizeof(unsigned char));
  convKernel<<<dimGridE, dimBlockE>>>(Nfil/2, Ncol/2, d_img1, d_out1, winSize, d_win1);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);
  
  hipSetDevice(1);
  hipMemcpyAsync(d_img2, h_img2, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
//  hipDeviceSetLimit(hipLimitMallocHeapSize, winSize * winSize * sizeof(unsigned char));
  convKernel<<<dimGridE, dimBlockE>>>(Nfil/2, Ncol/2, d_img2, d_out2, winSize, d_win2);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);

  hipSetDevice(2);
  hipMemcpyAsync(d_img3, h_img3, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
//  hipDeviceSetLimit(hipLimitMallocHeapSize, winSize * winSize * sizeof(unsigned char));
  convKernel<<<dimGridE, dimBlockE>>>(Nfil/2, Ncol/2, d_img3, d_out3, winSize, d_win3);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);

  hipSetDevice(3);
  hipMemcpyAsync(d_img4, h_img4, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
//  hipDeviceSetLimit(hipLimitMallocHeapSize, winSize * winSize * sizeof(unsigned char));
  convKernel<<<dimGridE, dimBlockE>>>(Nfil/2, Ncol/2, d_img4, d_out4, winSize, d_win4);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);
  hipSetDevice(0); hipDeviceSynchronize();
  hipSetDevice(1); hipDeviceSynchronize();
  hipSetDevice(2); hipDeviceSynchronize();
  hipSetDevice(3); hipDeviceSynchronize();

  printf("\n");
  printf("Kernel Elemento a Elemento\n");
  printf("Dimension problema: %d filas x %d columnas\n", Nfil, Ncol);
  printf("Dimension Block: %d x %d x %d (%d) threads\n", dimBlockE.x, dimBlockE.y, dimBlockE.z, dimBlockE.x * dimBlockE.y * dimBlockE.z);
  printf("Dimension Grid: %d x %d x %d (%d) blocks\n", dimGridE.x, dimGridE.y, dimGridE.z, dimGridE.x * dimGridE.y * dimGridE.z);

  // Ejecutar el kernel elemento a elemento
  //convKernel<<<dimGridE, dimBlockE>>>(Nfil, Ncol, d_img, d_out, 7);
  //CheckCudaError((char *) "Invocar Kernel", __LINE__);

  // Obtener el resultado desde el host 
  hipSetDevice(0);
  hipMemcpyAsync(h_out1, d_out1, numBytes, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);
  hipSetDevice(1);
  hipMemcpyAsync(h_out2, d_out2, numBytes, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);
  hipSetDevice(2);
  hipMemcpyAsync(h_out3, d_out3, numBytes, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);
  hipSetDevice(3);
  hipMemcpyAsync(h_out4, d_out4, numBytes, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

  for(int i = 0; i < 10; ++i){
    printf("%i ",h_out1[i]);
  }
  printf("\n");

  for(int i = 0; i < 10; ++i){
    printf("%i ",h_out2[i]);
  }
  printf("\n");
  for(int i = 0; i < 10; ++i){
    printf("%i ",h_out3[i]);
  }
  printf("\n");
  for(int i = 0; i < 10; ++i){
    printf("%i ",h_out4[i]);
  }
  printf("\n");
  // Guardar imagen
  for(int i = 0; i < imgSize/4; ++i){
    indCol = i%(Ncol/2);
    indFil = i/(Ncol/2);
    baseIndex = indCol + indFil * Ncol;
    // Abajo a la izquierda
    img[baseIndex*3] = h_out1[i];
    img[baseIndex*3+1] = h_out1[i];
    img[baseIndex*3+2] = h_out1[i];

    // Abajo a la derecha    
    img[(baseIndex+Ncol/2)*3] = h_out2[i];
    img[(baseIndex+Ncol/2)*3+1] = h_out2[i];
    img[(baseIndex+Ncol/2)*3+2] = h_out2[i];

    // Arriba a la izquierda    
    img[(baseIndex+Nfil/2*Ncol)*3] = h_out3[i];
    img[(baseIndex+Nfil/2*Ncol)*3+1] = h_out3[i];
    img[(baseIndex+Nfil/2*Ncol)*3+2] = h_out3[i];
    
    // Arriba a la derecha
    img[(baseIndex+Nfil/2*Ncol+Ncol/2)*3] = h_out4[i];
    img[(baseIndex+Nfil/2*Ncol+Ncol/2)*3+1] = h_out4[i];
    img[(baseIndex+Nfil/2*Ncol+Ncol/2)*3+2] = h_out4[i];
  }
  SaveBMP("out.bmp", h, img);

  // Liberar Memoria del device 
  hipSetDevice(0); hipFree(d_img1); hipFree(d_out1);
  hipSetDevice(1); hipFree(d_img2); hipFree(d_out2);
  hipSetDevice(2); hipFree(d_img3); hipFree(d_out3);
  hipSetDevice(3); hipFree(d_img4); hipFree(d_out4);

  hipDeviceSynchronize();

  // Imprimimos imagen original vs imagen resultante
 // printf("Imagen original:\n");
 // for (int i=0; i < 20 ; i++)
//	printf("%i\n", h_img[i]);
 
 // printf("Imagen resultante:\n");
  //for (int i=0; i < 20 ; i++)
//	printf("%i\n", h_out[i]);

  //printf("Imagen resultante:\n");
  //for (int i=0; i < 60 ; i++)
//	printf("%i\n", img[i]);
}

void InitM(int Nfil, int Ncol, int *M) {
  M[0] = 35;  
  M[1] = 35;  
  M[2] = 8;  
 
  M[3] = 35;  
  M[4] = 8;  
  M[5] = 35;  
 
  M[6] = 35;  
  M[7] = 35;  
  M[8] = 8;
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;

  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
  //else printf("(OK) %s \n", sms);
}
