#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <sys/resource.h>
extern "C" {
  #include "bmp.h"
}

#define THREADS 16

__global__ void convKernel(int Nfil, int Ncol, unsigned char *d_img, unsigned char *d_out, int win_size) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int ind = i * Ncol + j;
 
  unsigned char *win = (unsigned char* )malloc(win_size * win_size * sizeof(unsigned char));
  if(win == NULL) {
    d_out[ind] = -1;
    return;
  }

  if (i < Nfil && j < Ncol) {
    for(int win_i = -win_size/2; win_i <= win_size/2; win_i++) {
      for(int win_j = -win_size/2; win_j <= win_size/2; win_j++) {
        // Si esta fuera, extender el ultimo valor
        int img_i, img_j;
        if(i+win_i < 0)
          img_i = 0;
        else if(i+win_i >= Nfil)
           img_i = Nfil -1;
        else
           img_i = i + win_i;  
        
        if(j+win_j < 0)
          img_j = 0;
        else if(j+win_j >= Ncol)
           img_j = Ncol -1;
        else
           img_j = j + win_j;  

        win[(win_i+win_size/2) * win_size + win_j + win_size/2] = d_img[img_i * Ncol + img_j];
      } 
    }
    
  // Ordenar valores dentro de la ventana
    int swap;
    for (int c = 0 ; c < win_size * win_size - 1; c++) {
      for (int d = 0 ; d < win_size * win_size - c - 1; d++) {
        if (win[d] > win[d+1]) {
          swap   = win[d];
          win[d]   = win[d+1];
          win[d+1] = swap;
        }
      }
    }
   
   free(win);
   // Retornar la mediana
   if(ind < Nfil * Ncol)  d_out[ind] = win[win_size * win_size / 2];
  }
}


void InitM(int Nfil, int Ncol, int *v);
void CheckCudaError(char sms[], int line);


int main(int argc, char** argv) {
  unsigned int Nfil, Ncol;
  unsigned int numBytes;
  unsigned int nThreads;

  unsigned char *h_img, *h_out;
  unsigned char *d_img, *d_out;

  bmpInfoHeader* h = (bmpInfoHeader*) malloc(sizeof(bmpInfoHeader));
  unsigned char* img = LoadBMP("test.bmp", h);
  Nfil = h->height;
  Ncol = h->width;
  int imgSize = Ncol * Nfil;
  numBytes = imgSize * sizeof(unsigned char);
  
  h_img = (unsigned char*) malloc(numBytes);
  h_out = (unsigned char*) malloc(numBytes);
  
  if(h_img == NULL || h_out == NULL) {
    printf("Not enought memory to allocate image");
    exit(0);
  } 
 
  // Inicializar imagen
  for(int i = 0; i < imgSize; ++i){
    h_img[i] = img[i*3];
  }

  // Obtener Memoria en el device
  hipMalloc((unsigned char**)&d_img, numBytes); 
  hipMalloc((unsigned char**)&d_out, numBytes); 
  CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 

  // Copiar imagen desde el host en el device 
  hipMemcpy(d_img, h_img, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

  //
  // KERNEL ELEMENTO a ELEMENTO
  //

  nThreads = THREADS;
  int nBlocksFil = (Nfil+nThreads-1)/nThreads;
  int nBlocksCol = (Ncol+nThreads-1)/nThreads;

  dim3 dimGridE(nBlocksCol, nBlocksFil, 1);
  dim3 dimBlockE(nThreads, nThreads, 1);

  printf("\n");
  printf("Kernel Elemento a Elemento\n");
  printf("Dimension problema: %d filas x %d columnas\n", Nfil, Ncol);
  printf("Dimension Block: %d x %d x %d (%d) threads\n", dimBlockE.x, dimBlockE.y, dimBlockE.z, dimBlockE.x * dimBlockE.y * dimBlockE.z);
  printf("Dimension Grid: %d x %d x %d (%d) blocks\n", dimGridE.x, dimGridE.y, dimGridE.z, dimGridE.x * dimGridE.y * dimGridE.z);

  // Ejecutar el kernel elemento a elemento
  convKernel<<<dimGridE, dimBlockE>>>(Nfil, Ncol, d_img, d_out, 7);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);

  // Obtener el resultado desde el host 
  hipMemcpy(h_out, d_out, numBytes, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

  // Guardar imagen
  for(int i = 0; i < imgSize; ++i){
    img[i*3] = h_out[i];
    img[i*3+1] = h_out[i];
    img[i*3+2] = h_out[i];
  }
  SaveBMP("out.bmp", h, img);

  // Liberar Memoria del device 
  hipFree(d_img); hipFree(d_out);

  hipDeviceSynchronize();

  // Imprimimos imagen original vs imagen resultante
  printf("Imagen original:\n");
  for (int i=0; i < 20 ; i++)
	printf("%i\n", h_img[i]);
 
  printf("Imagen resultante:\n");
  for (int i=0; i < 20 ; i++)
	printf("%i\n", h_out[i]);

  printf("Imagen resultante:\n");
  for (int i=0; i < 60 ; i++)
	printf("%i\n", img[i]);
}

void InitM(int Nfil, int Ncol, int *M) {
  M[0] = 35;  
  M[1] = 35;  
  M[2] = 8;  
 
  M[3] = 35;  
  M[4] = 8;  
  M[5] = 35;  
 
  M[6] = 35;  
  M[7] = 35;  
  M[8] = 8;
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;

  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
  //else printf("(OK) %s \n", sms);
}
